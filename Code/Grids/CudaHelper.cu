#include "hip/hip_runtime.h"
#include "CudaHelper.h"

#pragma region cuda

__global__ void
LAUNCH_BOUND
_kernelReduceComp(hipDoubleComplex* arr, unsigned int uiJump, unsigned int uiMax)
{
    unsigned int uiIdFrom = (threadIdx.x + blockIdx.x * blockDim.x) * (uiJump << 1U) + uiJump;
    if (uiIdFrom < uiMax)
    {
        arr[uiIdFrom - uiJump] = hipCadd(arr[uiIdFrom - uiJump], arr[uiIdFrom]);
    }
}

#pragma endregion

static inline unsigned int GetReduceDim(unsigned int uiLength)
{
    unsigned int iRet = 0;
    while ((1U << iRet) < uiLength)
    {
        ++iRet;
    }
    return iRet;
}

hipDoubleComplex ReduceComplex(hipDoubleComplex* deviceBuffer, unsigned int uiLength)
{
    const unsigned int iRequiredDim = (uiLength + 1U) >> 1U;
    const unsigned int iPower = GetReduceDim(iRequiredDim);
    for (unsigned int i = 0; i <= iPower; ++i)
    {
        unsigned int iJump = 1U << i;
        unsigned int iThreadNeeded = 1U << (iPower - i);
        unsigned int iBlock = iThreadNeeded > _kMaxCudaThread ? iThreadNeeded / _kMaxCudaThread : 1U;
        unsigned int iThread = iThreadNeeded > _kMaxCudaThread ? _kMaxCudaThread : iThreadNeeded;
        _kernelReduceComp <<<iBlock, iThread >>> (deviceBuffer, iJump, uiLength);
    }
    hipDoubleComplex result[1];
    hipMemcpy(result, deviceBuffer, sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    return result[0];
}
