#include <hip/hip_runtime.h>
#include "../CGeneralGrids.h"
#include "../Integrator/CSparseGrid4D.h"
#include "../Logger.h"
#include "../CExtendedPath4D.h"
#include "Test1.h"

__device__ hipDoubleComplex test1_func1(const hipDoubleComplex& x, const hipDoubleComplex& y, const hipDoubleComplex& z, const hipDoubleComplex& w)
{
	hipDoubleComplex one = make_hipDoubleComplex(1.0, 0.0);

	return hipCdiv(one,
		hipCadd(hipCadd(x, hipCmul(y, y)),
			hipCadd(z, hipCmul(w, hipCmul(w, w)))));
}

__device__ hipDoubleComplex test1_func2(const hipDoubleComplex& x, const hipDoubleComplex& y, const hipDoubleComplex& z, const hipDoubleComplex& w)
{
	hipDoubleComplex one = make_hipDoubleComplex(1.0, 0.0);

	return hipCdiv(one,
		hipCadd(hipCadd(x, y),
			   hipCadd(z, w)));
}

__device__ integrand4d test1_fp1 = test1_func1;
__device__ integrand4d test1_fp2 = test1_func2;

void Test1()
{
	//CQuadratureReader reader("../Data/SparseGrid4D/GaussPatterson/", EIntegralDimension::EID_4D);
	CExtendedPath4D testgrid(5, 5, 1, "../Data/SparseGrid4D/GaussPatterson/");
	testgrid.SetIntegratorVerb(EVerboseLevel::GENERAL);
	integrand4d hfp1;
	hipMemcpyFromSymbol(&hfp1, HIP_SYMBOL(test1_fp1), sizeof(integrand4d));
	SIntegrateRes res = testgrid.Integrate(hfp1);
	LogGeneral("%d, res = %f + %f I", res.m_bDone, res.m_v.x, res.m_v.y);
	testgrid.PrintResoult("1/(x+y*y+z+w*w*w)");

	integrand4d hfp2;
	hipMemcpyFromSymbol(&hfp2, HIP_SYMBOL(test1_fp2), sizeof(integrand4d));
	res = testgrid.Integrate(hfp2);
	LogGeneral("%d, res = %f + %f I", res.m_bDone, res.m_v.x, res.m_v.y);
	testgrid.PrintResoult("1/(x+y+z+w)");
}


